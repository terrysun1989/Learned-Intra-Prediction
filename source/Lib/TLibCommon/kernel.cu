#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hipblas.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>

#include "TLibCommon/CommonDef.h"

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

extern "C" int top()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
} 

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>>(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

extern "C" int gemm(int const aW, int const aH, int const bW, int const bH, int const cW, int const cH, PRECISION *h_A, PRECISION *h_B, PRECISION *h_C)
{

	// 定义状态变量
	hipblasStatus_t status;

	/*
	** GPU 计算矩阵相乘
	*/

	// 创建并初始化 CUBLAS 库对象
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
			cout << "CUBLAS 对象实例化出错" << endl;
		}
		getchar();
		return EXIT_FAILURE;
	}

	PRECISION *d_A, *d_B, *d_C;
	// 在 显存 中为将要计算的矩阵开辟空间
	hipMalloc(
		(void**)&d_A,    // 指向开辟的空间的指针
		aW*aH * sizeof(PRECISION)    //　需要开辟空间的字节数
		);
	hipMalloc(
		(void**)&d_B,
		bW*bH * sizeof(PRECISION)
		);

	// 在 显存 中为将要存放运算结果的矩阵开辟空间
	hipMalloc(
		(void**)&d_C,
		cW*cH * sizeof(PRECISION)
		);

	// 将矩阵数据传递进 显存 中已经开辟好了的空间
	hipblasSetVector(
		aW*aH,    // 要存入显存的元素个数
		sizeof(PRECISION),    // 每个元素大小
		h_A,    // 主机端起始地址
		1,    // 连续元素之间的存储间隔
		d_A,    // GPU 端起始地址
		1    // 连续元素之间的存储间隔
		);
	hipblasSetVector(
		bW*bH,
		sizeof(PRECISION),
		h_B,
		1,
		d_B,
		1
		);

	// 同步函数
	hipDeviceSynchronize();

	//Timer myTimer;
	//myTimer.start();
	// 传递进矩阵相乘函数中的参数，具体含义请参考函数手册。
	PRECISION a = 1; PRECISION b = 0;
	// 矩阵相乘。该函数必然将数组解析成列优先数组
	hipblasDgemm(
		handle,    // blas 库对象
		HIPBLAS_OP_N,    // 矩阵 A 属性参数
		HIPBLAS_OP_N,    // 矩阵 B 属性参数
		bW,    // A, C 的行数
		aH,    // B, C 的列数
		bH,    // A 的列数和 B 的行数
		&a,    // 运算式的 α 值
		d_B,    // A 在显存中的地址
		bW,    // lda
		d_A,    // B 在显存中的地址
		aW,    // ldb
		&b,    // 运算式的 β 值
		d_C,    // C 在显存中的地址(结果矩阵)
		cW    // ldc
		);
	// 同步函数
	hipDeviceSynchronize();
	//myTimer.stop();
	//printf("use: %lf\n", myTimer.getElapsedTime());

	// 从 显存 中取出运算结果至 内存中去
	hipblasGetVector(
		cW*cH,    //  要取出元素的个数
		sizeof(PRECISION),    // 每个元素大小
		d_C,    // GPU 端起始地址
		1,    // 连续元素之间的存储间隔
		h_C,    // 主机端起始地址
		1    // 连续元素之间的存储间隔
		);


	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// 释放 CUBLAS 库对象
	hipblasDestroy(handle);


	//getchar();

	return 0;

}

